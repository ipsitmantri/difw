#include <hip/hip_runtime.h>

#include <iostream>
#include <float.h>

// FUNCTIONS

#define eps FLT_EPSILON;
#define inf INFINITY;

__device__ int sign(const int r){
    return (r > 0) - (r < 0);
}

__device__ int signf(const float r){
    return (r > 0) - (r < 0);
}

__device__ bool cmpf(float x, float y){
    return fabs(x - y) < eps;
}

__device__ bool cmpf0(const float& x){
    return fabs(x) < eps;
}

__device__ float right_boundary(const int& c, const float& xmin, const float& xmax, const int& nc){
    return xmin + (c + 1) * (xmax - xmin) / nc + eps;
}

__device__ float left_boundary(const int& c, const float& xmin, const float& xmax, const int& nc){
    return xmin + c * (xmax - xmin) / nc - eps;
}

__device__ int get_cell(const float& x, const float& xmin, const float& xmax, const int& nc){
    int c = floor((x - xmin) / (xmax - xmin) * nc);
    c = max(0, min(c, nc-1));
    return c;
}

__device__ float get_velocity(const float& x, const float* A, const int& n_batch, const int& batch_index, const float& xmin, const float& xmax, const int& nc){
    const int c = get_cell(x, xmin, xmax, nc);
    const float a = A[(2*c) * n_batch + batch_index];
    const float b = A[(2*c+1) * n_batch + batch_index];
    return a*x + b;
}

// INTEGRATION CLOSED FORM

__device__ float get_psi(const float& x, const float& t,  const float& a, const float& b){
    if (cmpf0(a)){
        return x + t*b;
    }
    else{
        return exp(t*a) * (x + (b/a)) - (b/a);
    }
}

__device__ float get_hit_time(float x, int c, const float& a, const float& b, const float& xmin, const float& xmax, const int& nc, float& xc, int& cc){

    const float v = a * x + b;
    if(cmpf0(v)) return inf;

    cc = c + signf(v);
    if(cc < 0 || cc >= nc) return inf;
    xc = (v > 0) ? right_boundary(c, xmin, xmax, nc) : left_boundary(c, xmin, xmax, nc);

    const float vc = a * xc + b;
    if(cmpf0(vc)) return inf;
    if(signf(v) != signf(vc)) return inf;
    if(xc == xmin || xc == xmax) return inf;

    if(cmpf0(a)){
        return (xc - x)/b;
    }else{
        return std::log(vc / v) / a;
    }
}

__device__ float integrate_closed_form(float x, float t, const float* A, const int& n_batch, const int& batch_index, const float& xmin, const float& xmax, const int& nc){
    int c = get_cell(x, xmin, xmax, nc);
    int cont = 0;
    const int contmax = std::max(c, nc-1-c);

    float a, b, xc, thit;
    int cc;
    while (true) {
        a = A[(2*c) * n_batch + batch_index];
        b = A[(2*c+1) * n_batch + batch_index];

        thit = get_hit_time(x, c, a, b, xmin, xmax, nc, xc, cc);
        if (thit > t){
            return get_psi(x, t, a, b);
        }

        x = xc;
        c = cc;
        t -= thit;

        cont++;
        if (cont > contmax){
            break;
        }
    }
    return -1;
}

__device__ void integrate_closed_form_trace(float* result, float x, float t, const float* A, const int& n_batch, const int& batch_index, const float& xmin, const float& xmax, const int& nc){
    int c = get_cell(x, xmin, xmax, nc);
    int cont = 0;
    const int contmax = std::max(c, nc-1-c);

    float a, b, xc, thit;
    int cc;
    while (true) {
        a = A[(2*c) * n_batch + batch_index];
        b = A[(2*c+1) * n_batch + batch_index];

        thit = get_hit_time(x, c, a, b, xmin, xmax, nc, xc, cc);
        if (thit > t){
            result[0] = get_psi(x, t, a, b);
            result[1] = t;
            result[2] = c;
            return;
        }

        x = xc;
        c = cc;
        t -= thit;

        cont++;
        if (cont > contmax){
            break;
        }
    }
    return;
}

// INTEGRATION NUMERIC

__device__ float get_psi_numeric(const float& x, const int& c, const float& t, const float* A, const int& n_batch, const int& batch_index){
    // const int c = get_cell(x, xmin, xmax, nc);
    const float a = A[(2*c) * n_batch + batch_index];
    const float b = A[(2*c+1) * n_batch + batch_index];
    if (cmpf0(a)){
        return x + t*b;
    }
    else{
        return std::exp(t*a) * (x + (b/a)) - (b/a);
    }
}

__device__ float get_phi_numeric(const float& x, const float& t, const int& nSteps2, const float* A, const int& n_batch, const int& batch_index, const float& xmin, const float& xmax, const int& nc){
    float yn = x;
    float midpoint;
    const float deltaT = t / nSteps2;
    for(int j = 0; j < nSteps2; j++) {
        midpoint = yn + deltaT / 2 * get_velocity(yn, A, n_batch, batch_index, xmin, xmax, nc);
        yn = yn + deltaT * get_velocity(midpoint, A, n_batch, batch_index, xmin, xmax, nc);
    }
    return yn;
}

__device__ float integrate_numeric(const float& x, const float& t, const float* A, const int& n_batch, const int& batch_index, const float& xmin, const float& xmax, const int& nc, const int& nSteps1, const int& nSteps2){
    float xPrev = x;
    const float deltaT = t / nSteps1;
    int c = get_cell(x, xmin, xmax, nc);
    for(int j = 0; j < nSteps1; j++) {
        float xTemp = get_psi_numeric(xPrev, c, deltaT, A, n_batch, batch_index);
        int cTemp = get_cell(xTemp, xmin, xmax, nc);
        if (c == cTemp){
            xPrev = xTemp;
        }
        else{
            xPrev = get_phi_numeric(xPrev, deltaT, nSteps2, A, n_batch, batch_index, xmin, xmax, nc);
            c = get_cell(xPrev, xmin, xmax, nc);
        }
    }
    return xPrev;
}


// DERIVATIVE

__device__ void derivative_psi_theta(double* gradpoints, const float& x, const int& c, const float& t, const int& d, const float* B, const float* A, const int& n_batch, const int& batch_index, const int& n_points, const int& point_index){
    const double a = A[(2*c) * n_batch + batch_index];
    const double b = A[(2*c+1) * n_batch + batch_index];

    
    if (cmpf0(a)){
        for(int k=0; k < d; k++){
            const double ak = B[(2*c)*d + k];
            const double bk = B[(2*c+1)*d + k];
            gradpoints[batch_index*(n_points * d) + point_index*d + k] += t*(x*ak + bk);
        }
    }
    else{
        const double tmp = exp(t*a);
        const double tmp1 = t * tmp * (x + b/a);
        const double tmp2 = (tmp-1)/pow(a, 2.0);
        for(int k=0; k < d; k++){
            const double ak = B[(2*c)*d + k];
            const double bk = B[(2*c+1)*d + k];
            gradpoints[batch_index*(n_points * d) + point_index*d + k] += ak * tmp1 + tmp2 * (bk*a - ak*b);
        }
    }
}

__device__ float derivative_phi_time(const float& x, const int& c, const float& t, const float* A, const int& n_batch, const int& batch_index){
    const double a = A[(2*c) * n_batch + batch_index];
    const double b = A[(2*c+1) * n_batch + batch_index];

    if (cmpf0(a)){
        return b;
    }
    else{
        return exp(t*a)*(a*x + b);
    }
}

__device__ void derivative_thit_theta(double* gradpoints, const float& x, const int& c, const float& xc, const int& d, const float* B, const float* A, const int& n_batch, const int& batch_index, const int& n_points, const int& point_index){
    const double a = A[(2*c) * n_batch + batch_index];
    const double b = A[(2*c+1) * n_batch + batch_index];

    if (cmpf0(a)){
        const double tmp = (x-xc) / pow(b, 2.0);
        for(int k=0; k < d; k++){
            const double bk = B[(2*c+1)*d + k];
            gradpoints[batch_index*(n_points * d) + point_index*d + k] -= tmp*bk;
        }
    }
    else{
        const double tmp1 = log( (a*xc + b) / (a*x + b) )/pow(a, 2.0);
        const double tmp2 = (x - xc) / (a * (a*x + b) * (a*xc + b) );
        for(int k=0; k < d; k++){
            const double ak = B[(2*c)*d + k];
            const double bk = B[(2*c+1)*d + k];

            const double d1 = - ak * tmp1;
            const double d2 = ( bk*a - ak*b) * tmp2;;
            gradpoints[batch_index*(n_points * d) + point_index*d + k] -= d1 + d2;
        }
    }
}

__device__ void derivative_phi_theta(double* gradpoints, const float& xini, const float& tm, const int& cm, const int& d, const float* B, const float* A, const int& n_batch, const int& batch_index, const int& n_points, const int& point_index, const float& xmin, const float& xmax, const int& nc){
    
    const int cini = get_cell(xini, xmin, xmax, nc);
    float xm = xini;

    if (cini != cm){
        float xc;
        const int step = sign(cm - cini);
        for (int c = cini; step*c < cm*step; c += step){
            if (step == 1){
                xc = right_boundary(c, xmin, xmax, nc);
            }else if (step == -1){
                xc = left_boundary(c, xmin, xmax, nc);
            }
            derivative_thit_theta(gradpoints, xm, c, xc, d, B, A, n_batch, batch_index, n_points, point_index);
            xm = xc;
        }
    }

    const float dpsi_dtime = derivative_phi_time(xm, cm, tm, A, n_batch, batch_index);
    for(int k=0; k < d; k++){
        gradpoints[batch_index*(n_points * d) + point_index*d + k] *= dpsi_dtime;
    }
    derivative_psi_theta(gradpoints, xm, cm, tm, d, B, A, n_batch, batch_index, n_points, point_index);
    
}


// KERNELS

__global__ void kernel_get_cell(
    const int n_points, const float* x, 
    const float xmin, const float xmax, const int nc, int* newpoints){  

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    if(point_index < n_points) {
        newpoints[point_index] = get_cell(x[point_index], xmin, xmax, nc);
    }
    return;
}

__global__ void kernel_get_velocity(
    const int n_points, const int n_batch, const float* x, const float* A, 
    const float xmin, const float xmax, const int nc, float* newpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    if(point_index < n_points && batch_index < n_batch) {
        newpoints[batch_index * n_points + point_index] = get_velocity(x[point_index], A, n_batch, batch_index, xmin, xmax, nc);
    }
    return;
}

__global__ void kernel_integrate_numeric(
    const int n_points, const int n_batch, const float* x, const float* A, 
    const float xmin, const float xmax, const int nc, 
    const int nSteps1, const int nSteps2, float* newpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    float t = 1.0;
    if(point_index < n_points && batch_index < n_batch) {
        newpoints[batch_index * n_points + point_index] = integrate_numeric(x[point_index], t, A, n_batch, batch_index, xmin, xmax, nc, nSteps1, nSteps2);
    }
    return;
}

__global__ void kernel_integrate_closed_form(
    const int n_points, const int n_batch, const float* x, const float* A, 
    const float xmin, const float xmax, const int nc, float* newpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    float t = 1.0;
    if(point_index < n_points && batch_index < n_batch) {
        newpoints[batch_index * n_points + point_index] = integrate_closed_form(x[point_index], t, A, n_batch, batch_index, xmin, xmax, nc);
    }
    return;
}

__global__ void kernel_derivative_closed_form(
    const int n_points, const int n_batch, const int d,
    const float* x, const float* A, const float* B, 
    const int xmin, const int xmax, const int nc, double* gradpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    
    float t = 1.0;
    const int e = 3;

    if(point_index < n_points && batch_index < n_batch){ 
        float result[e];
        integrate_closed_form_trace(result, x[point_index], t, A, n_batch, batch_index, xmin, xmax, nc);
            
        // float phi = result[0];
        float tm = result[1];
        int cm = result[2];
        derivative_phi_theta(gradpoints, x[point_index], tm, cm, d, B, A, n_batch, batch_index, n_points, point_index, xmin, xmax, nc);
        
    }
    return;
}

__global__ void kernel_integrate_closed_form_trace(
    const int n_points, const int n_batch, const float* x, const float* A, 
    const float xmin, const float xmax, const int nc, float* newpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    
    float t = 1.0;
    const int e = 3;

    if(point_index < n_points && batch_index < n_batch) {
        float result[e];
        integrate_closed_form_trace(result, x[point_index], t, A, n_batch, batch_index, xmin, xmax, nc);
        for(int p = 0; p < e; p++){
            newpoints[batch_index*(n_points * e) + point_index*e + p] = result[p];
        }
    }
    return;
}

__global__ void kernel_derivative_closed_form_trace(
    const int n_points, const int n_batch, const int d,
    const float* newpoints, const float* x, const float* A, const float* B, 
    const float xmin, const float xmax, const int nc, double* gradpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;

    const int e = 3;

    if(point_index < n_points && batch_index < n_batch) {
        // float phi = newpoints[batch_index*(n_points * e) + point_index*e + 0];
        float tm = newpoints[batch_index*(n_points * e) + point_index*e + 1];
        int cm = newpoints[batch_index*(n_points * e) + point_index*e + 2];
        
        derivative_phi_theta(gradpoints, x[point_index], tm, cm, d, B, A, n_batch, batch_index, n_points, point_index, xmin, xmax, nc);
    }
    return;
}